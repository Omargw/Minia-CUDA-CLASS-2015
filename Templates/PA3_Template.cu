#include "hip/hip_runtime.h"
/************************************************************/
/* Programming Assignment No. :   */
/* Team Members:											*/
/*				1-											*/
/*				2-											*/
/*															*/
/************************************************************/
/* "//@@" means you should insert your own code at current  */
/*  place													*/
/************************************************************/
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

#define TILE_WIDTH 16 

// Compute B = min_Element(A) , minimum in array A 
__global__ void min_Element(float * A, float * B) {
	//@@ Insert code to implement Minimum elemnt of array with reduction
	




	//
}

int main(void)
{
	int * hostA; // The A array
	int * hostB; // The B output , to store deviceB after parallel execution
	 
	int * B_check; // The output B matrix of CPU implementation) 
	int  * deviceA;
	int  * deviceB;
 
	const int width = 128;  // array width = 128 elemnts 
 
	 
	/* Allocating memory for input matrices HostA */
	hostA = new int[width];
	 

	/* Initializing input matrices with input random data */
	// Matrix A
	srand(1);
	for (int i = 0; i numARows; i++)
	{
		hostA[i] = 1 + (rand() % 100);  // genrate random between 1 and 100
	}
 
	//@@ Implement Min element of array in ordinary C++ code (serial code)
	//@@  Use it to check with the result of the parallel


	// print the result of the (Serial code) above
	printf("PUTPUT  SERAIL: %d \n", B_check);

	//@@ Allocate GPU memory here
	

	//@@ Copy memory to the GPU here
	

	//@@ Initialize the grid and block dimensions here
	

	//@@ Launch the GPU Kernel here



	hipDeviceSynchronize();
	//@@ Copy the GPU memory (deviceB) back to the CPU (hostB) here


	//@@ Free the GPU memory here


	 


	// print the result of the (Parallel code) above
	printf("PUTPUT  SERAIL: %d \n", hostB);
	
	

	return 0;
}