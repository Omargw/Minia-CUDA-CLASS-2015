#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;


#define Mask_width  5  // convolution kernel 
#define Mask_radius Mask_width/2  

//@@ INSERT CODE HERE
#define TILE_WIDTH 16

// CLAMP() In computer terms, to clamp a value is to make sure that it lies between some maximum and minimum values.
// If it�s greater than the max value, then it�s replaced by the max,
#define CLAMP(val, start, end) (min(max(val, start), end))

__global__ void conv2D(float *I, float *P, const float *__restrict__ M, int ImagW, int ImagH, int ImagC)
{
	 
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int row = by * TILE_WIDTH + ty;  // use of TILE_WIDTH instead of blockDim 
	int col = bx * TILE_WIDTH + tx;


	//@@ implement  your kernel here 
	 
    //@@ pixels are in the range of 0 to 1 , use CLAMP here at the end and before storing the data to M
		 
	}

	 


}

int main(int argc, char* argv[]) {
	 
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char * inputImageFile;
	char * inputMaskFile; 
	float * hostInputImageData;
	float * hostOutputImageData;
	float * hostMaskData;
	float * deviceInputImageData;
	float * deviceOutputImageData;
	float * deviceMaskData;

	 
    //@@ initialize your variables here

	 
	 
	//@@ INSERT CODE HERE
	dim3 dimGrid((imageWidth + TILE_WIDTH - 1) / TILE_WIDTH, (imageHeight + TILE_WIDTH - 1) / TILE_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, imageChannels);

	conv2D << <dimGrid, dimBlock >> >(deviceInputImageData, deviceOutputImageData, deviceMaskData, imageWidth, imageHeight, imageChannels);
	hipDeviceSynchronize();

	 


	 

	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);

	free(hostMaskData);
	 
	return 0;
}
