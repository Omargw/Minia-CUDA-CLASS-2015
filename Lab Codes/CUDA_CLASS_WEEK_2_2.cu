
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#define HEIGHT 5
#define WIDTH 5
#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16
#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(unsigned int)))
using namespace std;

__global__ void what_is_my_id_2d_A(unsigned int * const X, unsigned int * const Y, unsigned int * const thread)
{
	/*
	gridDim.x�  The size in blocks of the X dimension of the grid.
	gridDim.y�  The size in blocks of the Y dimension of the grid.
	blockDim.x� The size in threads of the X dimension of a single block.
	blockDim.y� The size in threads of the Y dimension of a single block.
	theadIdx.x� The offset within a block of the X thread index.
	theadIdx.y� The offset within a block of the Y thread index.
	*/
	const unsigned int idx_x  = (blockIdx.x * blockDim.x)  +  threadIdx.x;
	const unsigned int idx_y = (blockIdx.y * blockDim.y)  +  threadIdx.y;
	const unsigned int thread_idx =((gridDim.x * blockDim.x) * idx_y) + idx_x;
	X[thread_idx] = idx_x;

}

int main()
{



	int jimmy [HEIGHT][WIDTH];
	int n,m;


	for (n=0; n<HEIGHT; n++)
	{
		for (m=0; m<WIDTH; m++)
		{
			jimmy[n][m]=(n+1)*(m+1);
			cout <<  jimmy[n][m] << "  ";
		}
		cout << endl;
	}
	/**-------------------------------------------*/
	unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
	unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
	unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
	/* Total thread count � 32 * 4 = 128 */
	const dim3 threads_rect(32, 4); /* 32 * 4 */
	const dim3 blocks_rect(1,4);

	/* Total thread count � 16 * 8 = 128 */
	const dim3 threads_square(16, 8); /* 16 * 8 */
	const dim3 blocks_square(2,2);

	unsigned int * gpu_calc_thread;
	unsigned int * gpu_xthread;
	unsigned int * gpu_ythread;

	hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);

	for (int kernel=0; kernel < 2; kernel++)
	{
		switch (kernel)
		{
		case 0:
			{
				/* Execute our kernel */
				what_is_my_id_2d_A<<<blocks_rect, threads_rect>>>(gpu_xthread, gpu_ythread,
					gpu_calc_thread);
			} break;
		case 1:
			{
				/*Executeour kernel */
				what_is_my_id_2d_A<<<blocks_square, threads_square>>>(gpu_xthread, gpu_ythread,
					 gpu_calc_thread);
			} break;
		default: exit(1); break;
		}

		printf("\nKernel %d\n", kernel);
		/* Iterate through the arrays and print */
		for (int y=0; y < ARRAY_SIZE_Y; y++)
		{
			for (int x=0; x < ARRAY_SIZE_X; x++)
			{
				printf("CT:  %d TID: %d YTID: %d XTID: \n", cpu_calc_thread[y][x],  cpu_ythread[y][x], cpu_xthread[y][x]);  

			}
		}/* Wait for any key so we can see the console window */

		return 0;
	}
}

