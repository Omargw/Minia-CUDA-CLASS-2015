#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  // srand() , rand()
#include <stdio.h>
#include <math.h>


__global__ void global_reduce(int *input_d, int *output_d)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;

	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ int sdata[];
	sdata[tid] = input_d[i];
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[s + tid];
		}
		__syncthreads();
	}
	// only thread 0 writes result for this block back to global mem
	if (tid == 0){
		output_d[blockIdx.x] = sdata[0];
	}
}

int main()
{
	const int ARRAY_SIZE = 1 << 20;   
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
	const int maxThreadPerBlock = 1024;
	int threads = maxThreadPerBlock;
	int blocks = (ARRAY_SIZE / maxThreadPerBlock);


	int *input_h;
	input_h = (int*)malloc(ARRAY_BYTES);
	int sum = 0;
	srand(1);
	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		input_h[i] = (int)(rand() % 100);  // random number betwen 0 and 100
		sum += input_h[i];
	}
	printf("output SERAIL: %d \n", sum);
	// declare GPU memory pointers
	int *in_d, *intermediate_d, *out_d;

	// allocate GPU memory
	hipMalloc((void**)&in_d, ARRAY_BYTES);
	hipMalloc((void**)&intermediate_d, ARRAY_BYTES);
	hipMalloc((void**)&out_d, sizeof(int)); // only 1 element

	hipMemcpy(in_d, input_h, ARRAY_BYTES, hipMemcpyHostToDevice);

	global_reduce << <blocks, threads, threads * sizeof(int) >> >(in_d, intermediate_d);


	// now we're down to one block left, so reduce it
	threads = blocks; // launch one thread for each block in prev step
	blocks = 1;
	global_reduce << <blocks, threads, threads * sizeof(int) >> >(intermediate_d, out_d);

	int out_h;
	hipMemcpy(&out_h, out_d, sizeof(int), hipMemcpyDeviceToHost);

	printf("output parallel: %d \n", out_h);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	int cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	hipFree(in_d);
	hipFree(out_d);
	hipFree(intermediate_d);

	return 0;
}
