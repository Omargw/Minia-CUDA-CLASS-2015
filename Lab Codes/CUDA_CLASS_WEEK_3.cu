#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  // srand() , rand()
#include <stdio.h>
#include <math.h>
#include <algorithm> 
using namespace std;
__global__ void global_reduce(int *input_d, int *output_d)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	//int temp;
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			input_d[i] += input_d[s + i];
			/*temp = input_d[i];
			if (temp < input_d[s + i])
				input_d[i] = temp;*/
		}
		__syncthreads();
	}
	// only thread 0 writes result for this block back to global mem
	if (tid == 0){
		output_d[blockIdx.x] = input_d[i];
		//output_d[blockIdx.x] = input_d[i];
	}
}

int main()
{
	const int ARRAY_SIZE = 1 << 20;  // 1048576 Bytes = 8 Megabits
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
	const int maxThreadPerBlock = 1024;
	int threads = maxThreadPerBlock;
	int blocks =   (ARRAY_SIZE / maxThreadPerBlock);
    

	int *input_h;
	input_h = (int*)malloc(ARRAY_BYTES);
	int sum = 0;
	srand(1);
	int *min , temp;
	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		input_h[i] = (int)(rand() % 100);  // random number betwen 0 and 100
		sum += input_h[i];
		 
	}
	temp = input_h[0];
	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		 
		if (input_h[i] < temp)
			temp = input_h[i];
	}
	
	//printf("output SERAIL: %d \n", *min_element(input_h, input_h + ARRAY_SIZE-1));
	printf("output SERAIL: %d \n", sum);
	//printf("output SERAIL: %d , %d\n",input_h, input_h + 1);
	// declare GPU memory pointers
	int *in_d, *intermediate_d, *out_d;
	
	hipError_t err;

	// allocate GPU memory
	err = hipMalloc((void**)&in_d, ARRAY_BYTES);
	if (err != hipSuccess) {
		printf("error: cann't allocate.\n");
	
	}
	err = hipMalloc((void**)&intermediate_d ,ARRAY_BYTES);
	if (err != hipSuccess) {
		printf("error: cann't allocate.\n");

	}
	err = hipMalloc((void**)&out_d, sizeof(float)); // only 1 element
	if (err != hipSuccess) {
		printf("error: cann't allocate.\n");

	}

	hipMemcpy(in_d, input_h, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	global_reduce << <blocks, threads>> >(in_d, intermediate_d);
    

	// now we're down to one block left, so reduce it
	threads = blocks; // launch one thread for each block in prev step
	blocks = 1;
	global_reduce <<<blocks, threads >>>(intermediate_d, out_d);

	int out_h;
	hipMemcpy(&out_h, out_d, sizeof(int), hipMemcpyDeviceToHost);

	printf("output parallel: %d \n", out_h);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    int cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	hipFree(in_d);
	hipFree(out_d);
	hipFree(intermediate_d);
	
    return 0;
}
